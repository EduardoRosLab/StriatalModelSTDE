//https://gitlab.cern.ch/lhcb/Allen/blob/master/CMakeLists.txt


#include <hip/hip_runtime.h>
//#include <getopt.h>
//#include <iomanip>
#include <iostream>
//#include <vector>
//#include <algorithm>
using namespace std;

int main(int argc, char* argv[])
{
  int n_devices = 0;
  int rc = hipGetDeviceCount(&n_devices);
  if (rc != hipSuccess) {
    hipError_t error = hipGetLastError();
    std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
    return 0;
  }


  for (int cd = 0; cd < n_devices; ++cd) {
    hipDeviceProp_t dev;
    int rc = hipGetDeviceProperties(&dev, cd);
    if (rc != hipSuccess) {
      hipError_t error = hipGetLastError();
      std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
      return 0;
    }
    else {
      cout<<" arch=compute_"<<dev.major<<dev.minor<<",code=sm_"<<dev.major<<dev.minor;
    }
  }
  return n_devices;
}
