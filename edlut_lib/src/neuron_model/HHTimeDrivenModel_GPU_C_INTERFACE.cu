#include "hip/hip_runtime.h"
/***************************************************************************
 *                           HHTimeDrivenModel_GPU_C_INTERFACE.cu          *
 *                           -------------------                           *
 * copyright            : (C) 2015 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/HHTimeDrivenModel_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/HHTimeDrivenModel_GPU2.cuh"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/CurrentSynapseModel.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include "hip/hip_runtime.h"
#include ""

#include "integration_method/IntegrationMethodFactory_GPU_C_INTERFACE.cuh"

void HHTimeDrivenModel_GPU_C_INTERFACE::InitializeCurrentSynapsis(int N_neurons){
	this->CurrentSynapsis = new CurrentSynapseModel(N_neurons);
}

HHTimeDrivenModel_GPU_C_INTERFACE::HHTimeDrivenModel_GPU_C_INTERFACE(): TimeDrivenNeuronModel_GPU_C_INTERFACE(MilisecondScale), NeuronModel_GPU2(0),
	EXC(false), INH(false), NMDA(false), EXT_I(false){
	std::map<std::string, boost::any> param_map = HHTimeDrivenModel_GPU_C_INTERFACE::GetDefaultParameters();
	param_map["name"] = HHTimeDrivenModel_GPU_C_INTERFACE::GetName();
	this->SetParameters(param_map);

	this->State = (VectorNeuronState_GPU_C_INTERFACE *) new VectorNeuronState_GPU_C_INTERFACE(N_NeuronStateVariables);
}

HHTimeDrivenModel_GPU_C_INTERFACE::~HHTimeDrivenModel_GPU_C_INTERFACE(void){
	DeleteClassGPU2();
}

VectorNeuronState * HHTimeDrivenModel_GPU_C_INTERFACE::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * HHTimeDrivenModel_GPU_C_INTERFACE::ProcessInputSpike(Interconnection * inter, double time){
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] += inter->GetWeight();

	return 0;
}

void HHTimeDrivenModel_GPU_C_INTERFACE::ProcessInputCurrent(Interconnection * inter, Neuron * target, float current){
	//Update the external current in the corresponding input synapse of type EXT_I (defined in pA).
	this->CurrentSynapsis->SetInputCurrent(target->GetIndex_VectorNeuronState(), inter->GetSubindexType(), current);

	//Update the total external current that receive the neuron coming from all its EXT_I synapsis (defined in pA).
	float total_ext_I = this->CurrentSynapsis->GetTotalCurrent(target->GetIndex_VectorNeuronState());
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] = total_ext_I;
}


__global__ void HHTimeDrivenModel_GPU_C_INTERFACE_UpdateState(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}


bool HHTimeDrivenModel_GPU_C_INTERFACE::UpdateState(int index, double CurrentTime){
	if(prop.canMapHostMemory){
		HHTimeDrivenModel_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(State_GPU->AuxStateGPU,State_GPU->AuxStateCPU,this->N_TimeDependentNeuronState*State_GPU->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		HHTimeDrivenModel_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(State_GPU->InternalSpikeCPU,State_GPU->InternalSpikeGPU,State_GPU->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}

	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(State_GPU->VectorNeuronStates,State_GPU->VectorNeuronStates_GPU,State_GPU->GetNumberOfVariables()*State_GPU->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastUpdate,State_GPU->LastUpdateGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastSpikeTime,State_GPU->LastSpikeTimeGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	//The external current value it is not reset (N_TimeDependentNeuronState-1)
	memset(State_GPU->AuxStateCPU,0,(N_TimeDependentNeuronState-1)*State_GPU->SizeStates*sizeof(float));

	return false;
}

enum NeuronModelOutputActivityType HHTimeDrivenModel_GPU_C_INTERFACE::GetModelOutputActivityType(){
	return OUTPUT_SPIKE;
}

enum NeuronModelInputActivityType HHTimeDrivenModel_GPU_C_INTERFACE::GetModelInputActivityType(){
	return INPUT_SPIKE_AND_CURRENT;
}


ostream & HHTimeDrivenModel_GPU_C_INTERFACE::PrintInfo(ostream & out){
	out << "- HH Time-Driven Model GPU: " << HHTimeDrivenModel_GPU_C_INTERFACE::GetName() << endl;
	out << "\tExcitatory reversal potential (e_exc): " << this->e_exc << "mV" << endl;
	out << "\tInhibitory reversal potential (e_inh): " << this->e_inh << "mV" << endl;
	out << "\tEffective leak potential (e_leak): " << this->e_leak << "mV" << endl;
	out << "\tLeak conductance (g_leak): " << this->g_leak << "nS" << endl;
	out << "\tMembrane capacitance (c_m): " << this->c_m << "pF" << endl;
	out << "\tEffective threshold potential (v_thr): " << this->v_thr << "mV" << endl;
	out << "\tAMPA (excitatory) receptor time constant (tau_exc): " << this->tau_exc << "ms" << endl;
	out << "\tGABA (inhibitory) receptor time constant (tau_inh): " << this->tau_inh << "ms" << endl;
	out << "\tNMDA (excitatory) receptor time constant (tau_nmda): " << this->tau_nmda << "ms" << endl;
	out << "\tMaximum value of sodium conductance (g_na): " << this->g_na << "nS" << endl;
	out << "\tMaximum value of potassium conductance (g_kd): " << this->g_kd << "nS" << endl;
	out << "\tSodium potential (e_na): " << this->e_na << "mV" << endl;
	out << "\tPotassium potential (e_k): " << this->e_k << "mV" << endl;

	this->integration_method_GPU->PrintInfo(out);
	return out;
}


void HHTimeDrivenModel_GPU_C_INTERFACE::InitializeStates(int N_neurons, int OpenMPQueueIndex){

	//Select the correnpondent device.
	this->GPU_index = OpenMPQueueIndex % NumberOfGPUs;
	HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));

	this->State_GPU = (VectorNeuronState_GPU_C_INTERFACE *) this->State;

	//Initialize neural state variables.
	//m
	float alpha_m=0.32f*(13.0f-e_leak+v_thr)/(exp((13.0f-e_leak+v_thr)/4.0f)-1.0f);
	float beta_m=0.28f*(e_leak-v_thr-40.0f)/(exp((e_leak-v_thr-40.0f)/5.0f)-1.0f);
	float m_inf=alpha_m/(alpha_m+beta_m);

	//h
	float alpha_h=0.128f*exp((17.0f-e_leak+v_thr)/18.0f);
	float beta_h=4.0f/(1.0f+exp((40.0f-e_leak+v_thr)/5.0f));
	float h_inf=alpha_h/(alpha_h+beta_h);


	//n
	float alpha_n=0.032f*(15.0f-e_leak+v_thr)/(exp((15.0f-e_leak+v_thr)/5.0f)-1.0f);
	float beta_n=0.5f*exp((10.0f-e_leak+v_thr)/40.0f);
	float n_inf=alpha_n/(alpha_n+beta_n);

	float initialization[] = {e_leak, m_inf, h_inf, n_inf, 0.0f, 0.0f, 0.0f, 0.0f };

	State_GPU->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);

	InitializeVectorNeuronState_GPU2();

	//Initialize the array that stores the number of input current synapses for each neuron in the model
	InitializeCurrentSynapsis(N_neurons);
}




__global__ void HHTimeDrivenModel_GPU_C_INTERFACE_InitializeClassGPU2(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2,
		float new_e_exc, float new_e_inh, float new_e_leak, float new_g_leak, float new_c_m, float new_tau_exc,
		float new_tau_inh, float new_tau_nmda, float new_g_na, float new_g_kd, float new_e_na, float new_e_k, float new_v_thr,
		char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new HHTimeDrivenModel_GPU2(new_e_exc, new_e_inh, new_e_leak, new_g_leak, new_c_m, new_tau_exc,
				new_tau_inh, new_tau_nmda, new_g_na, new_g_kd, new_e_na, new_e_k, new_v_thr, integrationName, N_neurons, Buffer_GPU);
	}
}

void HHTimeDrivenModel_GPU_C_INTERFACE::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(HHTimeDrivenModel_GPU2 **));

	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
//REVISAR
	HANDLE_ERROR(hipMemcpy(integrationNameGPU, &integration_method_GPU->name[0], 32 * 4, hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integration_method_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	HHTimeDrivenModel_GPU_C_INTERFACE_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2, e_exc, e_inh, e_leak, g_leak, c_m, tau_exc,
		tau_inh, tau_nmda, g_na, g_kd, e_na, e_k, v_thr, integrationNameGPU, N_neurons, integration_method_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2, int NumberOfVariables, float * InitialStateGPU, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(NumberOfVariables, InitialStateGPU, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}

void HHTimeDrivenModel_GPU_C_INTERFACE::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU_C_INTERFACE *state = (VectorNeuronState_GPU_C_INTERFACE *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->NumberOfVariables, state->InitialStateGPU, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2);
	}
}


void HHTimeDrivenModel_GPU_C_INTERFACE::DeleteClassGPU2(){
	if (NeuronModel_GPU2 != 0){
		DeleteClass_GPU2 << <1, 1 >> >(NeuronModel_GPU2);
		hipFree(NeuronModel_GPU2);
	}
}


__global__ void SetEnabledSynapsis_GPU2(HHTimeDrivenModel_GPU2 ** NeuronModel_GPU2, bool new_EXC, bool new_INH, bool new_NMDA, bool new_EXT_I){
	if (blockIdx.x == 0 && threadIdx.x == 0){
		(*NeuronModel_GPU2)->SetEnabledSynapsis(new_EXC, new_INH, new_NMDA, new_EXT_I);
	}
}


bool HHTimeDrivenModel_GPU_C_INTERFACE::CheckSynapseType(Interconnection * connection){
	int Type = connection->GetType();
	if (Type<N_TimeDependentNeuronState && Type >= 0){
		//activaty synapse type
		if (Type == 0 && EXC == false){
			EXC = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 1 && INH == false){
			INH = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 2 && NMDA == false){
			NMDA = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 3 && EXT_I == false){
			EXT_I = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}

		NeuronModel * model = connection->GetSource()->GetNeuronModel();
		//Synapse types that process input spikes
		if (Type < N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_SPIKE){
				return true;
			}
			else{
			cout << "Synapses type " << Type << " of neuron model " << HHTimeDrivenModel_GPU_C_INTERFACE::GetName() << " must receive spikes. The source model generates currents." << endl;
				return false;
			}
		}
		//Synapse types that process input current
		if (Type == N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_CURRENT){
				connection->SetSubindexType(this->CurrentSynapsis->GetNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState()));
				this->CurrentSynapsis->IncrementNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState());
				return true;
			}
			else{
				cout << "Synapses type " << Type << " of neuron model " << HHTimeDrivenModel_GPU_C_INTERFACE::GetName() << " must receive current. The source model generates spikes." << endl;
				return false;
			}
		}
	}
	else{
		cout << "Neuron model " << HHTimeDrivenModel_GPU_C_INTERFACE::GetName() << " does not support input synapses of type " << Type << ". Just defined " << N_TimeDependentNeuronState << " synapses types." << endl;
		return false;
	}
}

std::map<std::string, boost::any> HHTimeDrivenModel_GPU_C_INTERFACE::GetParameters() const {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetParameters();
	newMap["e_exc"] = boost::any(this->e_exc); //Excitatory reversal potential (mV)
	newMap["e_inh"] = boost::any(this->e_inh); // Inhibitory reversal potential mV)
	newMap["e_leak"] = boost::any(this->e_leak); // Effective leak potential(mV)
	newMap["g_leak"] = boost::any(this->g_leak); // Leak conductance (nS)
	newMap["c_m"] = boost::any(this->c_m); // Membrane capacitance (pF)
	newMap["v_thr"] = boost::any(this->v_thr); // Effective threshold potential (mV)
	newMap["tau_exc"] = boost::any(this->tau_exc); // AMPA (excitatory) receptor time constant (ms)
	newMap["tau_inh"] = boost::any(this->tau_inh); // GABA (inhibitory) receptor time constant (ms)
	newMap["tau_nmda"] = boost::any(this->tau_nmda); // NMDA (excitatory) receptor time constant (ms)
	newMap["g_na"] = boost::any(this->g_na); // Maximum value of sodium conductance (nS)
	newMap["g_kd"] = boost::any(this->g_kd); // Maximum value of potassium conductance (nS)
	newMap["e_na"] = boost::any(this->e_na); // Sodium potential (mV)
	newMap["e_k"] = boost::any(this->e_k); // Potassium potential (mV)
	return newMap;
}

std::map<std::string, boost::any> HHTimeDrivenModel_GPU_C_INTERFACE::GetSpecificNeuronParameters(int index) const throw (EDLUTException){
	return GetParameters();
}

void HHTimeDrivenModel_GPU_C_INTERFACE::SetParameters(std::map<std::string, boost::any> param_map) throw (EDLUTException){

	// Search for the parameters in the dictionary
	std::map<std::string,boost::any>::iterator it=param_map.find("e_exc");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_exc = new_param;
		param_map.erase(it);
	}

	it=param_map.find("e_inh");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_inh = new_param;
		param_map.erase(it);
	}

	it=param_map.find("e_leak");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_leak = new_param;
		param_map.erase(it);
	}

	it=param_map.find("g_leak");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->g_leak = new_param;
		param_map.erase(it);
	}

	it=param_map.find("c_m");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->c_m = new_param;
		param_map.erase(it);
	}

	it = param_map.find("v_thr");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->v_thr = new_param;
		param_map.erase(it);
	}

	it=param_map.find("tau_exc");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_exc = new_param;
		param_map.erase(it);
	}

	it=param_map.find("tau_inh");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_inh = new_param;
		param_map.erase(it);
	}

	it=param_map.find("tau_nmda");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_nmda = new_param;
		param_map.erase(it);
	}

	it=param_map.find("g_na");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->g_na = new_param;
		param_map.erase(it);
	}

	it=param_map.find("g_kd");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->g_kd = new_param;
		param_map.erase(it);
	}

	it=param_map.find("e_na");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_na = new_param;
		param_map.erase(it);
	}

	it=param_map.find("e_k");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_k = new_param;
		param_map.erase(it);
	}

	// Search for the parameters in the dictionary
	TimeDrivenNeuronModel_GPU_C_INTERFACE::SetParameters(param_map);

	return;
}


IntegrationMethod_GPU_C_INTERFACE * HHTimeDrivenModel_GPU_C_INTERFACE::CreateIntegrationMethod(ModelDescription imethodDescription) throw (EDLUTException){
	return IntegrationMethodFactory_GPU_C_INTERFACE<HHTimeDrivenModel_GPU_C_INTERFACE>::CreateIntegrationMethod_GPU(imethodDescription, (HHTimeDrivenModel_GPU_C_INTERFACE*) this);
}


std::map<std::string, boost::any> HHTimeDrivenModel_GPU_C_INTERFACE::GetDefaultParameters() {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetDefaultParameters<HHTimeDrivenModel_GPU_C_INTERFACE>();
	newMap["e_exc"] = boost::any(0.0f); //Excitatory reversal potential (mV)
	newMap["e_inh"] = boost::any(-80.0f); // Inhibitory reversal potential mV)
	newMap["e_leak"] = boost::any(-65.0f); // Effective leak potential(mV)
	newMap["g_leak"] = boost::any(10.0f); // Leak conductance (nS)
	newMap["c_m"] = boost::any(120.0f); // Membrane capacitance (pF)
	newMap["v_thr"] = boost::any(-52.0f); // Effective threshold potential (mV)
	newMap["tau_exc"] = boost::any(5.0f); // AMPA (excitatory) receptor time constant (ms)
	newMap["tau_inh"] = boost::any(10.0f); // GABA (inhibitory) receptor time constant (ms)
	newMap["tau_nmda"] = boost::any(20.0f); // NMDA (excitatory) receptor time constant (ms)
	newMap["g_na"] = boost::any(20000.0f); // Maximum value of sodium conductance (nS)
	newMap["g_kd"] = boost::any(6000.0f); // Maximum value of potassium conductance (nS)
	newMap["e_na"] = boost::any(50.0f); // Sodium potential (mV)
	newMap["e_k"] = boost::any(-90.0f); // Potassium potential (mV)
	return newMap;
}

NeuronModel* HHTimeDrivenModel_GPU_C_INTERFACE::CreateNeuronModel(ModelDescription nmDescription){
	HHTimeDrivenModel_GPU_C_INTERFACE * nmodel = new HHTimeDrivenModel_GPU_C_INTERFACE();
	nmodel->SetParameters(nmDescription.param_map);
	return nmodel;
}

ModelDescription HHTimeDrivenModel_GPU_C_INTERFACE::ParseNeuronModel(std::string FileName) throw (EDLUTFileException){
	FILE *fh;
	ModelDescription nmodel;
	nmodel.model_name = HHTimeDrivenModel_GPU_C_INTERFACE::GetName();
	long Currentline = 0L;
	fh=fopen(FileName.c_str(),"rt");
	if(!fh) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_LOAD, ERROR_NEURON_MODEL_OPEN, REPAIR_NEURON_MODEL_NAME, Currentline, FileName.c_str());
	}

	Currentline = 1L;
	float param;

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_E_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_E_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_E_LEAK, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_leak"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_G_LEAK, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["g_leak"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_C_M, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["c_m"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_V_THR, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["v_thr"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_TAU_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_TAU_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_TAU_NMDA, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_nmda"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_G_NA, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["g_na"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_G_KD, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["g_kd"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_E_NA, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_na"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_HH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_HH_TIME_DRIVEN_MODEL_E_K, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_k"] = boost::any(param);

	skip_comments(fh, Currentline);
	try {
		ModelDescription intMethodDescription = TimeDrivenNeuronModel_GPU_C_INTERFACE::ParseIntegrationMethod<HHTimeDrivenModel_GPU_C_INTERFACE>(fh, Currentline);
		nmodel.param_map["int_meth"] = boost::any(intMethodDescription);
	}
	catch (EDLUTException exc) {
		throw EDLUTFileException(exc, Currentline, FileName.c_str());
	}

	nmodel.param_map["name"] = boost::any(HHTimeDrivenModel_GPU_C_INTERFACE::GetName());

	fclose(fh);

	return nmodel;
}

std::string HHTimeDrivenModel_GPU_C_INTERFACE::GetName(){
	return "HHTimeDrivenModel_GPU";
}

std::map<std::string, std::string> HHTimeDrivenModel_GPU_C_INTERFACE::GetNeuronModelInfo() {
	// Return a dictionary with the parameters
	std::map<std::string, std::string> newMap;
	newMap["info"] = std::string("GPU Time-driven Hodgkin and Huxley (HH) neuron model with four differential equations (membrane potential (v) and three ionic-channel variables (m, h and n)) and four types of input synapses: AMPA (excitatory), GABA (inhibitory), NMDA (excitatory) and external input current (set on pA)");
	newMap["e_exc"] = std::string("Excitatory reversal potential (mV)");
	newMap["e_inh"] = std::string("Inhibitory reversal potential mV)");
	newMap["e_leak"] = std::string("Effective leak potential(mV)");
	newMap["g_leak"] = std::string("Leak conductance (nS)");
	newMap["c_m"] = std::string("Membrane capacitance (pF)");
	newMap["v_thr"] = std::string("Effective threshold potential (mV)");
	newMap["tau_exc"] = std::string("AMPA (excitatory) receptor time constant (ms)");
	newMap["tau_inh"] = std::string("GABA (inhibitory) receptor time constant (ms)");
	newMap["tau_nmda"] = std::string("NMDA (excitatory) receptor time constant (ms)");
	newMap["g_na"] = std::string("Maximum value of sodium conductance (nS)");
	newMap["g_kd"] = std::string("Maximum value of potassium conductance (nS)");
	newMap["e_na"] = std::string("Sodium potential (mV)");
	newMap["e_k"] = std::string("Potassium potential (mV)");
	newMap["int_meth"] = std::string("Integraton method dictionary (from the list of available integration methods in GPU)");

	return newMap;
}
