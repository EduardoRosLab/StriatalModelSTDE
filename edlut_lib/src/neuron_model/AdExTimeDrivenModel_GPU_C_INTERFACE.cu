#include "hip/hip_runtime.h"
/***************************************************************************
 *                           AdExTimeDrivenModel_GPU_C_INTERFACE.cu        *
 *                           -------------------                           *
 * copyright            : (C) 2015 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/AdExTimeDrivenModel_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/AdExTimeDrivenModel_GPU2.cuh"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/CurrentSynapseModel.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include "hip/hip_runtime.h"
#include ""

#include "integration_method/IntegrationMethodFactory_GPU_C_INTERFACE.cuh"

void AdExTimeDrivenModel_GPU_C_INTERFACE::InitializeCurrentSynapsis(int N_neurons){
	this->CurrentSynapsis = new CurrentSynapseModel(N_neurons);
}

AdExTimeDrivenModel_GPU_C_INTERFACE::AdExTimeDrivenModel_GPU_C_INTERFACE(): TimeDrivenNeuronModel_GPU_C_INTERFACE(MilisecondScale), a(0), b(0),
	thr_slo_fac(0), v_thr(0), tau_w(0), e_exc(0), e_inh(0), e_reset(0), e_leak(0), g_leak(0), c_m(0), tau_exc(0), tau_inh(0), NeuronModel_GPU2(0),
	EXC(false), INH(false), NMDA(false), EXT_I(false){

	std::map<std::string, boost::any> param_map = AdExTimeDrivenModel_GPU_C_INTERFACE::GetDefaultParameters();
	param_map["name"] = AdExTimeDrivenModel_GPU_C_INTERFACE::GetName();
	this->SetParameters(param_map);

	this->State = (VectorNeuronState_GPU_C_INTERFACE *) new VectorNeuronState_GPU_C_INTERFACE(N_NeuronStateVariables);
}


AdExTimeDrivenModel_GPU_C_INTERFACE::~AdExTimeDrivenModel_GPU_C_INTERFACE(void){
	DeleteClassGPU2();
}

VectorNeuronState * AdExTimeDrivenModel_GPU_C_INTERFACE::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * AdExTimeDrivenModel_GPU_C_INTERFACE::ProcessInputSpike(Interconnection * inter, double time){
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] += inter->GetWeight();

	return 0;
}

void AdExTimeDrivenModel_GPU_C_INTERFACE::ProcessInputCurrent(Interconnection * inter, Neuron * target, float current){
	//Update the external current in the corresponding input synapse of type EXT_I (defined in pA).
	this->CurrentSynapsis->SetInputCurrent(target->GetIndex_VectorNeuronState(), inter->GetSubindexType(), current);

	//Update the total external current that receive the neuron coming from all its EXT_I synapsis (defined in pA).
	float total_ext_I = this->CurrentSynapsis->GetTotalCurrent(target->GetIndex_VectorNeuronState());
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] = total_ext_I;
}


__global__ void AdExTimeDrivenModel_GPU_C_INTERFACE_UpdateState(AdExTimeDrivenModel_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}


bool AdExTimeDrivenModel_GPU_C_INTERFACE::UpdateState(int index, double CurrentTime){
	if(prop.canMapHostMemory){
		AdExTimeDrivenModel_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(State_GPU->AuxStateGPU,State_GPU->AuxStateCPU,this->N_TimeDependentNeuronState*State_GPU->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		AdExTimeDrivenModel_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(State_GPU->InternalSpikeCPU,State_GPU->InternalSpikeGPU,State_GPU->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}

	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(State_GPU->VectorNeuronStates,State_GPU->VectorNeuronStates_GPU,State_GPU->GetNumberOfVariables()*State_GPU->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastUpdate,State_GPU->LastUpdateGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastSpikeTime,State_GPU->LastSpikeTimeGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	//The external current value it is not reset (N_TimeDependentNeuronState-1)
	memset(State_GPU->AuxStateCPU,0,(N_TimeDependentNeuronState-1)*State_GPU->SizeStates*sizeof(float));

	return false;
}


enum NeuronModelOutputActivityType AdExTimeDrivenModel_GPU_C_INTERFACE::GetModelOutputActivityType(){
	return OUTPUT_SPIKE;
}

enum NeuronModelInputActivityType AdExTimeDrivenModel_GPU_C_INTERFACE::GetModelInputActivityType(){
	return INPUT_SPIKE_AND_CURRENT;
}


ostream & AdExTimeDrivenModel_GPU_C_INTERFACE::PrintInfo(ostream & out){
	out << "- AdEx Time-Driven Model GPU: " << AdExTimeDrivenModel_GPU_C_INTERFACE::GetName() << endl;
	out << "\tConductance (a): " << this->a << "nS" << endl;
	out << "\tSpike trigger adaptation (b): " << this->b << "pA" << endl;
	out << "\tThreshold slope factor (thr_slo_fac): " << this->thr_slo_fac << "mV" << endl;
	out << "\tEffective threshold potential (v_thr): " << this->v_thr << "mV" << endl;
	out << "\tAdaptation time constant (tau_w): " << this->tau_w << "ms" << endl;
	out << "\tExcitatory reversal potential (e_exc): " << this->e_exc << "mV" << endl;
	out << "\tInhibitory reversal potential (e_inh): " << this->e_inh << "mV" << endl;
	out << "\tReset potential (e_reset): " << this->e_reset << "mV" << endl;
	out << "\tEffective leak potential (e_leak): " << this->e_leak << "mV" << endl;
	out << "\tLeak conductance (g_leak): " << this->g_leak << "nS" << endl;
	out << "\tMembrane capacitance (c_m): " << this->c_m << "pF" << endl;
	out << "\tAMPA (excitatory) receptor time constant (tau_exc): " << this->tau_exc << "ms" << endl;
	out << "\tGABA (inhibitory) receptor time constant (tau_inh): " << this->tau_inh << "ms" << endl;
	out << "\tNMDA (excitatory) receptor time constant (tau_nmda): " << this->tau_nmda << "ms" << endl;

	this->integration_method_GPU->PrintInfo(out);
	return out;
}


void AdExTimeDrivenModel_GPU_C_INTERFACE::InitializeStates(int N_neurons, int OpenMPQueueIndex){

	//Select the correnpondent device.
	this->GPU_index = OpenMPQueueIndex % NumberOfGPUs;
	HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));

	this->State_GPU = (VectorNeuronState_GPU_C_INTERFACE *) this->State;

	//Initialize neural state variables.
	float initialization[] = { e_leak, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
	State_GPU->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);

	InitializeVectorNeuronState_GPU2();

	//Initialize the array that stores the number of input current synapses for each neuron in the model
	InitializeCurrentSynapsis(N_neurons);
}



__global__ void AdExTimeDrivenModel_GPU_C_INTERFACE_InitializeClassGPU2(AdExTimeDrivenModel_GPU2 ** NeuronModel_GPU2, float a, float b, float thr_slo_fac,
	float v_thr, float tau_w, float e_exc, float e_inh, float e_reset, float e_leak, float g_leak, float c_m, float tau_exc, float tau_inh, float tau_nmda,
		char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new AdExTimeDrivenModel_GPU2(a, b, thr_slo_fac, v_thr, tau_w, e_exc, e_inh, e_reset, e_leak, g_leak, c_m, tau_exc, tau_inh, tau_nmda,
			integrationName, N_neurons, Buffer_GPU);
	}
}

void AdExTimeDrivenModel_GPU_C_INTERFACE::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(AdExTimeDrivenModel_GPU2 **));

	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
//REVISAR
	HANDLE_ERROR(hipMemcpy(integrationNameGPU, &integration_method_GPU->name[0], 32 * 4, hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integration_method_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	AdExTimeDrivenModel_GPU_C_INTERFACE_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2, a, b, thr_slo_fac, v_thr, tau_w, e_exc, e_inh, e_reset, e_leak, g_leak, c_m,
		tau_exc, tau_inh, tau_nmda, integrationNameGPU, N_neurons, integration_method_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}


__global__ void initializeVectorNeuronState_GPU2(AdExTimeDrivenModel_GPU2 ** NeuronModel_GPU2, int NumberOfVariables, float * InitialStateGPU, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(NumberOfVariables, InitialStateGPU, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}


void AdExTimeDrivenModel_GPU_C_INTERFACE::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU_C_INTERFACE *state = (VectorNeuronState_GPU_C_INTERFACE *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->NumberOfVariables, state->InitialStateGPU, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(AdExTimeDrivenModel_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2);
	}
}


void AdExTimeDrivenModel_GPU_C_INTERFACE::DeleteClassGPU2(){
	if (NeuronModel_GPU2 != 0){
		DeleteClass_GPU2 << <1, 1 >> >(NeuronModel_GPU2);
		hipFree(NeuronModel_GPU2);
	}
}


__global__ void SetEnabledSynapsis_GPU2(AdExTimeDrivenModel_GPU2 ** NeuronModel_GPU2, bool new_EXC, bool new_INH, bool new_NMDA, bool new_EXT_I){
	if (blockIdx.x == 0 && threadIdx.x == 0){
		(*NeuronModel_GPU2)->SetEnabledSynapsis(new_EXC, new_INH, new_NMDA, new_EXT_I);
	}
}


bool AdExTimeDrivenModel_GPU_C_INTERFACE::CheckSynapseType(Interconnection * connection){
	int Type = connection->GetType();
	if (Type<N_TimeDependentNeuronState && Type >= 0){
		//activaty synapse type
		if (Type == 0 && EXC == false){
			EXC = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 1 && INH == false){
			INH = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 2 && NMDA == false){
			NMDA = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 3 && EXT_I == false){
			EXT_I = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}

		NeuronModel * model = connection->GetSource()->GetNeuronModel();
		//Synapse types that process input spikes
		if (Type < N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_SPIKE){
				return true;
			}
			else{
			cout << "Synapses type " << Type << " of neuron model " << AdExTimeDrivenModel_GPU_C_INTERFACE::GetName() << " must receive spikes. The source model generates currents." << endl;
				return false;
			}
		}
		//Synapse types that process input current
		if (Type == N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_CURRENT){
				connection->SetSubindexType(this->CurrentSynapsis->GetNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState()));
				this->CurrentSynapsis->IncrementNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState());
				return true;
			}
			else{
				cout << "Synapses type " << Type << " of neuron model " << AdExTimeDrivenModel_GPU_C_INTERFACE::GetName() << " must receive current. The source model generates spikes." << endl;
				return false;
			}
		}
	}
	else{
		cout << "Neuron model " << AdExTimeDrivenModel_GPU_C_INTERFACE::GetName() << " does not support input synapses of type " << Type << ". Just defined " << N_TimeDependentNeuronState << " synapses types." << endl;
		return false;
	}
}

std::map<std::string, boost::any> AdExTimeDrivenModel_GPU_C_INTERFACE::GetParameters() const {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetParameters();
	newMap["a"] = boost::any(this->a); //conductance (nS)
	newMap["b"] = boost::any(this->b); //spike trigger adaptation (pA)
	newMap["thr_slo_fac"] = boost::any(this->thr_slo_fac); //threshold slope factor (mV)
	newMap["v_thr"] = boost::any(this->v_thr); //effective threshold potential (mV)
	newMap["tau_w"] = boost::any(this->tau_w); //adaptation time constant (ms)
	newMap["e_exc"] = boost::any(this->e_exc); //excitatory reversal potential (mV)
	newMap["e_inh"] = boost::any(this->e_inh); //inhibitory reversal potential (mV)
	newMap["e_reset"] = boost::any(this->e_reset); //reset potential (mV)
	newMap["e_leak"] = boost::any(this->e_leak); //effective leak potential (mV)
	newMap["g_leak"] = boost::any(this->g_leak); //leak conductance (nS)
	newMap["c_m"] = boost::any(this->c_m); //membrane capacitance (pF)
	newMap["tau_exc"] = boost::any(this->tau_exc); //AMPA (excitatory) receptor time constant (ms)
	newMap["tau_inh"] = boost::any(this->tau_inh); //GABA (inhibitory) receptor time constant (ms)
	newMap["tau_nmda"] = boost::any(this->tau_nmda); //NMDA (excitatory) receptor time constant (ms)
	return newMap;
}

std::map<std::string, boost::any> AdExTimeDrivenModel_GPU_C_INTERFACE::GetSpecificNeuronParameters(int index) const throw (EDLUTException){
	return GetParameters();
}

void AdExTimeDrivenModel_GPU_C_INTERFACE::SetParameters(std::map<std::string, boost::any> param_map) throw (EDLUTException){

	// Search for the parameters in the dictionary
	std::map<std::string, boost::any>::iterator it = param_map.find("a");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->a = new_param;
		param_map.erase(it);
	}

	it = param_map.find("b");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->b = new_param;
		param_map.erase(it);
	}

	it = param_map.find("thr_slo_fac");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->thr_slo_fac = new_param;
		param_map.erase(it);
	}

	it = param_map.find("v_thr");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->v_thr = new_param;
		param_map.erase(it);
	}

	it = param_map.find("tau_w");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_w = new_param;
		param_map.erase(it);
	}

	it = param_map.find("e_exc");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_exc = new_param;
		param_map.erase(it);
	}

	it = param_map.find("e_inh");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_inh = new_param;
		param_map.erase(it);
	}

	it = param_map.find("e_reset");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_reset = new_param;
		param_map.erase(it);
	}

	it = param_map.find("e_leak");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_leak = new_param;
		param_map.erase(it);
	}

	it = param_map.find("g_leak");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->g_leak = new_param;
		param_map.erase(it);
	}

	it = param_map.find("c_m");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->c_m = new_param;
		param_map.erase(it);
	}

	it = param_map.find("tau_exc");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_exc = new_param;
		param_map.erase(it);
	}

	it = param_map.find("tau_inh");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_inh = new_param;
		param_map.erase(it);
	}


	it=param_map.find("tau_nmda");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_nmda = new_param;
		param_map.erase(it);
	}

	// Search for the parameters in the dictionary
	TimeDrivenNeuronModel_GPU_C_INTERFACE::SetParameters(param_map);
	return;
}


IntegrationMethod_GPU_C_INTERFACE * AdExTimeDrivenModel_GPU_C_INTERFACE::CreateIntegrationMethod(ModelDescription imethodDescription) throw (EDLUTException){
	return IntegrationMethodFactory_GPU_C_INTERFACE<AdExTimeDrivenModel_GPU_C_INTERFACE>::CreateIntegrationMethod_GPU(imethodDescription, (AdExTimeDrivenModel_GPU_C_INTERFACE*) this);
}


std::map<std::string, boost::any> AdExTimeDrivenModel_GPU_C_INTERFACE::GetDefaultParameters() {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetDefaultParameters<AdExTimeDrivenModel_GPU_C_INTERFACE>();
	newMap["a"] = boost::any(1.0f); //conductance (nS)
	newMap["b"] = boost::any(9.0f); //spike trigger adaptation (pA)
	newMap["thr_slo_fac"] = boost::any(2.0f); //threshold slope factor (mV)
	newMap["v_thr"] = boost::any(-50.0f); //effective threshold potential (mV)
	newMap["tau_w"] = boost::any(50.0f); //adaptation time constant (ms)
	newMap["e_exc"] = boost::any(0.0f); //excitatory reversal potential (mV)
	newMap["e_inh"] = boost::any(-80.0f); //inhibitory reversal potential (mV)
	newMap["e_reset"] = boost::any(-80.0f); //reset potential (mV)
	newMap["e_leak"] = boost::any(-65.0f); //effective leak potential (mV)
	newMap["g_leak"] = boost::any(10.0f); //leak conductance (nS)
	newMap["c_m"] = boost::any(110.0f); //membrane capacitance (pF)
	newMap["tau_exc"] = boost::any(5.0f); //AMPA (excitatory) receptor time constant (ms)
	newMap["tau_inh"] = boost::any(10.0f); //GABA (inhibitory) receptor time constant (ms)
	newMap["tau_nmda"] = boost::any(20.0f); //NMDA (excitatory) receptor time constant (ms)
	return newMap;
}

NeuronModel* AdExTimeDrivenModel_GPU_C_INTERFACE::CreateNeuronModel(ModelDescription nmDescription){
	AdExTimeDrivenModel_GPU_C_INTERFACE * nmodel = new AdExTimeDrivenModel_GPU_C_INTERFACE();
	nmodel->SetParameters(nmDescription.param_map);
	return nmodel;
}

ModelDescription AdExTimeDrivenModel_GPU_C_INTERFACE::ParseNeuronModel(std::string FileName) throw (EDLUTFileException){
	FILE *fh;
	ModelDescription nmodel;
	nmodel.model_name = AdExTimeDrivenModel_GPU_C_INTERFACE::GetName();
	long Currentline = 0L;
	fh = fopen(FileName.c_str(), "rt");
	if (!fh) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_NEURON_MODEL_OPEN, REPAIR_NEURON_MODEL_NAME, Currentline, FileName.c_str());
	}

	Currentline = 1L;
	skip_comments(fh, Currentline);

	float param;
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_A, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["a"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_B, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["b"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_THR_SLO_FAC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["thr_slo_fac"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_V_THR, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["v_thr"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_TAU_W, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_w"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_E_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_E_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_E_RESET, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_reset"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_E_LEAK, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_leak"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_G_LEAK, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["g_leak"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_C_M, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["c_m"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_TAU_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_TAU_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_ADEX_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_ADEX_TIME_DRIVEN_MODEL_TAU_NMDA, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_nmda"] = boost::any(param);

	skip_comments(fh, Currentline);
	try {
		ModelDescription intMethodDescription = TimeDrivenNeuronModel_GPU_C_INTERFACE::ParseIntegrationMethod<AdExTimeDrivenModel_GPU_C_INTERFACE>(fh, Currentline);
		nmodel.param_map["int_meth"] = boost::any(intMethodDescription);
	}
	catch (EDLUTException exc) {
		throw EDLUTFileException(exc, Currentline, FileName.c_str());
	}

	nmodel.param_map["name"] = boost::any(AdExTimeDrivenModel_GPU_C_INTERFACE::GetName());

	fclose(fh);

	return nmodel;
}

std::string AdExTimeDrivenModel_GPU_C_INTERFACE::GetName(){
	return "AdExTimeDrivenModel_GPU";
}

std::map<std::string, std::string> AdExTimeDrivenModel_GPU_C_INTERFACE::GetNeuronModelInfo() {
	// Return a dictionary with the parameters
	std::map<std::string, std::string> newMap;
	newMap["info"] = std::string("GPU Time-driven Adaptative Exponential Integrate and Fire (AdEx) neuron model with two differential equations(membrane potential (v) and membrane recovery (w)) and four types of input synapses: AMPA (excitatory), GABA (inhibitory), NMDA (excitatory) and external input current (set on pA)");
	newMap["a"] = std::string("Conductance (nS)");
	newMap["b"] = std::string("Spike trigger adaptation (pA)");
	newMap["thr_slo_fac"] = std::string("Threshold slope factor (mV)");
	newMap["v_thr"] = std::string("Effective threshold potential (mV)");
	newMap["tau_w"] = std::string("Adaptation time constant (ms)");
	newMap["e_exc"] = std::string("Excitatory reversal potential (mV)");
	newMap["e_inh"] = std::string("Inhibitory reversal potential (mV)");
	newMap["e_reset"] = std::string("Reset potential (mV)");
	newMap["e_leak"] = std::string("Effective leak potential (mV)");
	newMap["g_leak"] = std::string("Leak conductance (nS)");
	newMap["c_m"] = std::string("Membrane capacitance (pF)");
	newMap["tau_exc"] = std::string("AMPA (excitatory) receptor time constant (ms)");
	newMap["tau_inh"] = std::string("GABA (inhibitory) receptor time constant (ms)");
	newMap["tau_nmda"] = std::string("NMDA (excitatory) receptor time constant (ms)");
	newMap["int_meth"] = std::string("Integraton method dictionary (from the list of available integration methods in GPU)");

	return newMap;
}
