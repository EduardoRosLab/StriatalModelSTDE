#include "hip/hip_runtime.h"
/***************************************************************************
 *                           IzhikevichTimeDrivenModel_GPU_C_INTERFACE.cu  *
 *                           -------------------                           *
 * copyright            : (C) 2015 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/IzhikevichTimeDrivenModel_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/IzhikevichTimeDrivenModel_GPU2.cuh"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/CurrentSynapseModel.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include "hip/hip_runtime.h"
#include ""

#include "integration_method/IntegrationMethodFactory_GPU_C_INTERFACE.cuh"

void IzhikevichTimeDrivenModel_GPU_C_INTERFACE::InitializeCurrentSynapsis(int N_neurons){
	this->CurrentSynapsis = new CurrentSynapseModel(N_neurons);
}

IzhikevichTimeDrivenModel_GPU_C_INTERFACE::IzhikevichTimeDrivenModel_GPU_C_INTERFACE(): TimeDrivenNeuronModel_GPU_C_INTERFACE(MilisecondScale), a(0.0), b(0), c(0),
d(0), e_exc(0), e_inh(0), c_m(0), tau_exc(0), tau_inh(0), tau_nmda(0), NeuronModel_GPU2(0), EXC(false), INH(false), NMDA(false), EXT_I(false){

	std::map<std::string, boost::any> param_map = IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetDefaultParameters();
	param_map["name"] = IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetName();
	this->SetParameters(param_map);

	this->State = (VectorNeuronState_GPU_C_INTERFACE *) new VectorNeuronState_GPU_C_INTERFACE(N_NeuronStateVariables);
}


IzhikevichTimeDrivenModel_GPU_C_INTERFACE::~IzhikevichTimeDrivenModel_GPU_C_INTERFACE(void){
	DeleteClassGPU2();
}

VectorNeuronState * IzhikevichTimeDrivenModel_GPU_C_INTERFACE::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * IzhikevichTimeDrivenModel_GPU_C_INTERFACE::ProcessInputSpike(Interconnection * inter, double time){
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] += inter->GetWeight();

	return 0;
}

void IzhikevichTimeDrivenModel_GPU_C_INTERFACE::ProcessInputCurrent(Interconnection * inter, Neuron * target, float current){
	//Update the external current in the corresponding input synapse of type EXT_I (defined in pA).
	this->CurrentSynapsis->SetInputCurrent(target->GetIndex_VectorNeuronState(), inter->GetSubindexType(), current);

	//Update the total external current that receive the neuron coming from all its EXT_I synapsis (defined in pA).
	float total_ext_I = this->CurrentSynapsis->GetTotalCurrent(target->GetIndex_VectorNeuronState());
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] = total_ext_I;
}


__global__ void IzhikevichTimeDrivenModel_GPU_C_INTERFACE_UpdateState(IzhikevichTimeDrivenModel_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}


bool IzhikevichTimeDrivenModel_GPU_C_INTERFACE::UpdateState(int index, double CurrentTime){
	if(prop.canMapHostMemory){
		IzhikevichTimeDrivenModel_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(State_GPU->AuxStateGPU,State_GPU->AuxStateCPU,this->N_TimeDependentNeuronState*State_GPU->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		IzhikevichTimeDrivenModel_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(State_GPU->InternalSpikeCPU,State_GPU->InternalSpikeGPU,State_GPU->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}

	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(State_GPU->VectorNeuronStates,State_GPU->VectorNeuronStates_GPU,State_GPU->GetNumberOfVariables()*State_GPU->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastUpdate,State_GPU->LastUpdateGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastSpikeTime,State_GPU->LastSpikeTimeGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	//The external current value it is not reset (N_TimeDependentNeuronState-1)
	memset(State_GPU->AuxStateCPU,0,(N_TimeDependentNeuronState-1)*State_GPU->SizeStates*sizeof(float));

	return false;
}


enum NeuronModelOutputActivityType IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetModelOutputActivityType(){
	return OUTPUT_SPIKE;
}

enum NeuronModelInputActivityType IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetModelInputActivityType(){
	return INPUT_SPIKE_AND_CURRENT;
}


ostream & IzhikevichTimeDrivenModel_GPU_C_INTERFACE::PrintInfo(ostream & out){
	out << "- Izhikevich Time-Driven Model GPU: " << IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetName() << endl;
	out << "\tTime scale of recovery variable u (a): " << this->a << "dimensionless" << endl;
	out << "\tSensitivity of the recovery variable u to the subthreshold fluctuations of the membrane potential v (b): " << this->b << "dimensionless" << endl;
	out << "\tAfter-spike reset value of the membrane potential v (c): " << this->c << "dimensionless" << endl;
	out << "\tAfter-spike reset of the recovery variable u (d): " << this->d << "dimensionless" << endl;
	out << "\tExcitatory reversal potential (e_exc): " << this->e_exc << "mV" << endl;
	out << "\tInhibitory reversal potential (e_inh): " << this->e_inh << "mV" << endl;
	out << "\tMembrane capacitance (c_m): " << this->c_m << "pF" << endl;
	out << "\tAMPA (excitatory) receptor time constant (tau_exc): " << this->tau_exc << "ms" << endl;
	out << "\tGABA (inhibitory) receptor time constant (tau_inh): " << this->tau_inh << "ms" << endl;
	out << "\tNMDA (excitatory) receptor time constant (tau_nmda): " << this->tau_nmda << "ms" << endl;

	this->integration_method_GPU->PrintInfo(out);
	return out;
}


void IzhikevichTimeDrivenModel_GPU_C_INTERFACE::InitializeStates(int N_neurons, int OpenMPQueueIndex){

	//Select the correnpondent device.
	this->GPU_index = OpenMPQueueIndex % NumberOfGPUs;
	HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));

	this->State_GPU = (VectorNeuronState_GPU_C_INTERFACE *) this->State;

	//Initialize neural state variables.
	float Veq=(((b-5)-sqrt((5-b)*(5-b)-22.4f))/0.08f);
	float Ueq=Veq*b;

	float initialization[] = {Veq, Ueq,0.0f,0.0f,0.0f,0.0f};
	State_GPU->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);

	InitializeVectorNeuronState_GPU2();

	//Initialize the array that stores the number of input current synapses for each neuron in the model
	InitializeCurrentSynapsis(N_neurons);
}



__global__ void IzhikevichTimeDrivenModel_GPU_C_INTERFACE_InitializeClassGPU2(IzhikevichTimeDrivenModel_GPU2 ** NeuronModel_GPU2,
		float a, float b, float c, float d, float e_exc, float e_inh, float c_m, float tau_exc, float tau_inh, float tau_nmda,
		char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new IzhikevichTimeDrivenModel_GPU2(a, b, c, d, e_exc, e_inh, c_m, tau_exc, tau_inh, tau_nmda, integrationName,
			N_neurons, Buffer_GPU);
	}
}

void IzhikevichTimeDrivenModel_GPU_C_INTERFACE::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(IzhikevichTimeDrivenModel_GPU2 **));

	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
//REVISAR
	HANDLE_ERROR(hipMemcpy(integrationNameGPU, &integration_method_GPU->name[0], 32 * 4, hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integration_method_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	IzhikevichTimeDrivenModel_GPU_C_INTERFACE_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2, a, b, c, d, e_exc, e_inh,
		c_m, tau_exc, tau_inh, tau_nmda, integrationNameGPU, N_neurons, integration_method_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(IzhikevichTimeDrivenModel_GPU2 ** NeuronModel_GPU2, int NumberOfVariables, float * InitialStateGPU, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(NumberOfVariables, InitialStateGPU, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}


void IzhikevichTimeDrivenModel_GPU_C_INTERFACE::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU_C_INTERFACE *state = (VectorNeuronState_GPU_C_INTERFACE *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->NumberOfVariables, state->InitialStateGPU, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(IzhikevichTimeDrivenModel_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2);
	}
}


void IzhikevichTimeDrivenModel_GPU_C_INTERFACE::DeleteClassGPU2(){
	if (NeuronModel_GPU2 != 0){
		DeleteClass_GPU2 << <1, 1 >> >(NeuronModel_GPU2);
		hipFree(NeuronModel_GPU2);
	}
}


__global__ void SetEnabledSynapsis_GPU2(IzhikevichTimeDrivenModel_GPU2 ** NeuronModel_GPU2, bool new_EXC, bool new_INH, bool new_NMDA, bool new_EXT_I){
	if (blockIdx.x == 0 && threadIdx.x == 0){
		(*NeuronModel_GPU2)->SetEnabledSynapsis(new_EXC, new_INH, new_NMDA, new_EXT_I);
	}
}


bool IzhikevichTimeDrivenModel_GPU_C_INTERFACE::CheckSynapseType(Interconnection * connection){
	int Type = connection->GetType();
	if (Type<N_TimeDependentNeuronState && Type >= 0){
		//activaty synapse type
		if (Type == 0 && EXC == false){
			EXC = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 1 && INH == false){
			INH = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 2 && NMDA == false){
			NMDA = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 3 && EXT_I == false){
			EXT_I = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}

		NeuronModel * model = connection->GetSource()->GetNeuronModel();
		//Synapse types that process input spikes
		if (Type < N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_SPIKE){
				return true;
			}
			else{
			cout << "Synapses type " << Type << " of neuron model " << IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetName() << " must receive spikes. The source model generates currents." << endl;
				return false;
			}
		}
		//Synapse types that process input current
		if (Type == N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_CURRENT){
				connection->SetSubindexType(this->CurrentSynapsis->GetNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState()));
				this->CurrentSynapsis->IncrementNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState());
				return true;
			}
			else{
				cout << "Synapses type " << Type << " of neuron model " << IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetName() << " must receive current. The source model generates spikes." << endl;
				return false;
			}
		}
	}
	else{
		cout << "Neuron model " << IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetName() << " does not support input synapses of type " << Type << ". Just defined " << N_TimeDependentNeuronState << " synapses types." << endl;
		return false;
	}
}

std::map<std::string, boost::any> IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetParameters() const {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetParameters();
	newMap["a"] = boost::any(this->a); // Time scale of recovery variable u (dimensionless)
	newMap["b"] = boost::any(this->b); // Sensitivity of the recovery variable u to the subthreshold fluctuations of the membrane potential v (dimensionless)
	newMap["c"] = boost::any(this->c);  // After-spike reset value of the membrane potential v (dimensionless)
	newMap["d"] = boost::any(this->d); // After-spike reset of the recovery variable u (dimensionless)
	newMap["e_exc"] = boost::any(this->e_exc); // Excitatory reversal potential (mV)
	newMap["e_inh"] = boost::any(this->e_inh); // Inhibitory reversal potential (mV)
	newMap["c_m"] = boost::any(this->c_m); // Membrane capacitance (pF)
	newMap["tau_exc"] = boost::any(this->tau_exc); // AMPA (excitatory) receptor time constant (ms)
	newMap["tau_inh"] = boost::any(this->tau_inh); // GABA (inhibitory) receptor time constant (ms)
	newMap["tau_nmda"] = boost::any(this->tau_nmda); // NMDA (excitatory) receptor time constant (ms)
	return newMap;
}

std::map<std::string, boost::any> IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetSpecificNeuronParameters(int index) const throw (EDLUTException){
	return GetParameters();
}

void IzhikevichTimeDrivenModel_GPU_C_INTERFACE::SetParameters(std::map<std::string, boost::any> param_map) throw (EDLUTException){

	// Search for the parameters in the dictionary
	std::map<std::string, boost::any>::iterator it = param_map.find("a");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->a = new_param;
		param_map.erase(it);
	}

	it = param_map.find("b");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->b = new_param;
		param_map.erase(it);
	}

	it = param_map.find("c");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->c = new_param;
		param_map.erase(it);
	}

	it = param_map.find("d");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->d = new_param;
		param_map.erase(it);
	}

	it = param_map.find("e_exc");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_exc = new_param;
		param_map.erase(it);
	}

	it = param_map.find("e_inh");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_inh = new_param;
		param_map.erase(it);
	}

	it = param_map.find("c_m");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->c_m = new_param;
		param_map.erase(it);
	}

	it = param_map.find("tau_exc");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_exc = new_param;
		param_map.erase(it);
	}

	it = param_map.find("tau_inh");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_inh = new_param;
		param_map.erase(it);
	}


	it=param_map.find("tau_nmda");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_nmda = new_param;
		param_map.erase(it);
	}

	// Search for the parameters in the dictionary
	TimeDrivenNeuronModel_GPU_C_INTERFACE::SetParameters(param_map);
	return;
}


IntegrationMethod_GPU_C_INTERFACE * IzhikevichTimeDrivenModel_GPU_C_INTERFACE::CreateIntegrationMethod(ModelDescription imethodDescription) throw (EDLUTException){
	return IntegrationMethodFactory_GPU_C_INTERFACE<IzhikevichTimeDrivenModel_GPU_C_INTERFACE>::CreateIntegrationMethod_GPU(imethodDescription, (IzhikevichTimeDrivenModel_GPU_C_INTERFACE*) this);
}


std::map<std::string, boost::any> IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetDefaultParameters() {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetDefaultParameters<IzhikevichTimeDrivenModel_GPU_C_INTERFACE>();
	newMap["a"] = boost::any(0.1f); // Time scale of recovery variable u (dimensionless)
	newMap["b"] = boost::any(0.23f); // Sensitivity of the recovery variable u to the subthreshold fluctuations of the membrane potential v (dimensionless)
	newMap["c"] = boost::any(-65.0f); // After-spike reset value of the membrane potential v (dimensionless)
	newMap["d"] = boost::any(0.2f); // After-spike reset of the recovery variable u (dimensionless)
	newMap["e_exc"] = boost::any(0.0f); // Excitatory reversal potential (mV)
	newMap["e_inh"] = boost::any(-80.0f); // Inhibitory reversal potential (mV)
	newMap["c_m"] = boost::any(100.0f); // Membrane capacitance (pF)
	newMap["tau_exc"] = boost::any(5.0f); // AMPA (excitatory) receptor time constant (ms)
	newMap["tau_inh"] = boost::any(10.0f); // GABA (inhibitory) receptor time constant (ms)
	newMap["tau_nmda"] = boost::any(20.0f); // NMDA (excitatory) receptor time constant (ms)
	return newMap;
}

NeuronModel* IzhikevichTimeDrivenModel_GPU_C_INTERFACE::CreateNeuronModel(ModelDescription nmDescription){
	IzhikevichTimeDrivenModel_GPU_C_INTERFACE * nmodel = new IzhikevichTimeDrivenModel_GPU_C_INTERFACE();
	nmodel->SetParameters(nmDescription.param_map);
	return nmodel;
}

ModelDescription IzhikevichTimeDrivenModel_GPU_C_INTERFACE::ParseNeuronModel(std::string FileName) throw (EDLUTFileException){
	FILE *fh;
	ModelDescription nmodel;
	nmodel.model_name = IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetName();
	long Currentline = 0L;
	fh=fopen(FileName.c_str(),"rt");
	if(!fh) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_NEURON_MODEL_OPEN, REPAIR_NEURON_MODEL_NAME, Currentline, FileName.c_str());
	}

	Currentline = 1L;
	skip_comments(fh, Currentline);

	float param;
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_A, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["a"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_B, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["b"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_C, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["c"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_D, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["d"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_E_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_E_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_C_M, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["c_m"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_TAU_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_TAU_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_IZHIKEVICH_TIME_DRIVEN_MODEL_GPU_LOAD, ERROR_IZHIKEVICH_TIME_DRIVEN_MODEL_TAU_NMDA, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_nmda"] = boost::any(param);

	skip_comments(fh, Currentline);
	try {
		ModelDescription intMethodDescription = TimeDrivenNeuronModel_GPU_C_INTERFACE::ParseIntegrationMethod<IzhikevichTimeDrivenModel_GPU_C_INTERFACE>(fh, Currentline);
		nmodel.param_map["int_meth"] = boost::any(intMethodDescription);
	}
	catch (EDLUTException exc) {
		throw EDLUTFileException(exc, Currentline, FileName.c_str());
	}

	nmodel.param_map["name"] = boost::any(IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetName());

	fclose(fh);

	return nmodel;
}

std::string IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetName(){
	return "IzhikevichTimeDrivenModel_GPU";
}

std::map<std::string, std::string> IzhikevichTimeDrivenModel_GPU_C_INTERFACE::GetNeuronModelInfo() {
	// Return a dictionary with the parameters
	std::map<std::string, std::string> newMap;
	newMap["info"] = std::string("GPU Time-driven Izhikevich (Izhikevich) neuron model with two differential equations(membrane potential (v) and membrane recovery (u)) and four types of input synapses: AMPA (excitatory), GABA (inhibitory), NMDA (excitatory) and external input current (set on pA)");
	newMap["a"] = std::string("Time scale of recovery variable u (dimensionless)");
	newMap["b"] = std::string("Sensitivity of the recovery variable u to the subthreshold fluctuations of the membrane potential v (dimensionless)");
	newMap["c"] = std::string("After-spike reset value of the membrane potential v (dimensionless)");
	newMap["d"] = std::string("After-spike reset of the recovery variable u (dimensionless)");
	newMap["e_exc"] = std::string("Excitatory reversal potential (mV)");
	newMap["e_inh"] = std::string("Inhibitory reversal potential (mV)");
	newMap["c_m"] = std::string("Membrane capacitance (pF)");
	newMap["tau_exc"] = std::string("AMPA (excitatory) receptor time constant (ms)");
	newMap["tau_inh"] = std::string("GABA (inhibitory) receptor time constant (ms)");
	newMap["tau_nmda"] = std::string("NMDA (excitatory) receptor time constant (ms)");
	newMap["int_meth"] = std::string("Integraton method dictionary (from the list of available integration methods in GPU)");

	return newMap;
}