#include "hip/hip_runtime.h"
/***************************************************************************
 *                           LIFTimeDrivenModel_IS_GPU_C_INTERFACE.cu      *
 *                           -------------------                           *
 * copyright            : (C) 2015 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/LIFTimeDrivenModel_IS_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/LIFTimeDrivenModel_IS_GPU2.cuh"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/CurrentSynapseModel.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include "hip/hip_runtime.h"
#include ""

#include "integration_method/IntegrationMethodFactory_GPU_C_INTERFACE.cuh"

void LIFTimeDrivenModel_IS_GPU_C_INTERFACE::InitializeCurrentSynapsis(int N_neurons){
	this->CurrentSynapsis = new CurrentSynapseModel(N_neurons);
}

LIFTimeDrivenModel_IS_GPU_C_INTERFACE::LIFTimeDrivenModel_IS_GPU_C_INTERFACE() : TimeDrivenNeuronModel_GPU_C_INTERFACE(SecondScale), e_exc(0), e_inh(0), e_leak(0), c_m(0), v_thr(0), tau_exc(0), tau_inh(0),
tau_ref(0), g_leak(0), tau_nmda(0), NeuronModel_GPU2(0), EXC(false), INH(false), NMDA(false), EXT_I(false){
	std::map<std::string, boost::any> param_map = LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetDefaultParameters();
	param_map["name"] = LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetName();
	this->SetParameters(param_map);

	this->State = (VectorNeuronState_GPU_C_INTERFACE *) new VectorNeuronState_GPU_C_INTERFACE(N_NeuronStateVariables);
}


LIFTimeDrivenModel_IS_GPU_C_INTERFACE::~LIFTimeDrivenModel_IS_GPU_C_INTERFACE(void){
	DeleteClassGPU2();
}

VectorNeuronState * LIFTimeDrivenModel_IS_GPU_C_INTERFACE::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * LIFTimeDrivenModel_IS_GPU_C_INTERFACE::ProcessInputSpike(Interconnection * inter, double time){
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] += inter->GetWeight();

	return 0;
}

void LIFTimeDrivenModel_IS_GPU_C_INTERFACE::ProcessInputCurrent(Interconnection * inter, Neuron * target, float current){
	//Update the external current in the corresponding input synapse of type EXT_I (defined in pA).
	this->CurrentSynapsis->SetInputCurrent(target->GetIndex_VectorNeuronState(), inter->GetSubindexType(), current);

	//Update the total external current that receive the neuron coming from all its EXT_I synapsis (defined in pA).
	float total_ext_I = this->CurrentSynapsis->GetTotalCurrent(target->GetIndex_VectorNeuronState());
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] = total_ext_I;
}


__global__ void LIFTimeDrivenModel_IS_GPU_C_INTERFACE_UpdateState(LIFTimeDrivenModel_IS_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}


bool LIFTimeDrivenModel_IS_GPU_C_INTERFACE::UpdateState(int index, double CurrentTime){
	if(prop.canMapHostMemory){
		LIFTimeDrivenModel_IS_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(State_GPU->AuxStateGPU,State_GPU->AuxStateCPU,this->N_TimeDependentNeuronState*State_GPU->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		LIFTimeDrivenModel_IS_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(State_GPU->InternalSpikeCPU,State_GPU->InternalSpikeGPU,State_GPU->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}

	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(State_GPU->VectorNeuronStates,State_GPU->VectorNeuronStates_GPU,State_GPU->GetNumberOfVariables()*State_GPU->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastUpdate,State_GPU->LastUpdateGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastSpikeTime,State_GPU->LastSpikeTimeGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	//The external current value it is not reset (N_TimeDependentNeuronState-1)
	memset(State_GPU->AuxStateCPU,0,(N_TimeDependentNeuronState-1)*State_GPU->SizeStates*sizeof(float));

	return false;
}


enum NeuronModelOutputActivityType LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetModelOutputActivityType(){
	return OUTPUT_SPIKE;
}

enum NeuronModelInputActivityType LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetModelInputActivityType(){
	return INPUT_SPIKE_AND_CURRENT;
}


ostream & LIFTimeDrivenModel_IS_GPU_C_INTERFACE::PrintInfo(ostream & out){
	out << "- Leaky Time-Driven Model GPU: " << LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetName() << endl;
	out << "\tExcitatory reversal potential (e_exc): " << this->e_exc << "V" << endl;
	out << "\tInhibitory reversal potential (e_inh): " << this->e_inh << "V" << endl;
	out << "\tEffective leak potential (e_leak): " << this->e_leak << "V" << endl;
	out << "\tMembrane capacitance (c_m): " << this->c_m << "F" << endl;
	out << "\tEffective threshold potential (v_thr): " << this->v_thr << "V" << endl;
	out << "\tAMPA (excitatory) receptor time constant (tau_exc): " << this->tau_exc << "s" << endl;
	out << "\tGABA (inhibitory) receptor time constant (tau_inh): " << this->tau_inh << "s" << endl;
	out << "\tRefractory period (tau_ref): " << this->tau_ref << "s" << endl;
	out << "\tLeak conductance (g_leak): " << this->g_leak << "S" << endl;
	out << "\tNMDA (excitatory) receptor time constant (tau_nmda): " << this->tau_nmda << "s" << endl;

	this->integration_method_GPU->PrintInfo(out);
	return out;
}


void LIFTimeDrivenModel_IS_GPU_C_INTERFACE::InitializeStates(int N_neurons, int OpenMPQueueIndex){
	//Select the correnpondent device.
	this->GPU_index = OpenMPQueueIndex % NumberOfGPUs;
	HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));

	this->State_GPU = (VectorNeuronState_GPU_C_INTERFACE *) this->State;

	//Initialize neural state variables.
	float initialization[] = {e_leak,0.0f,0.0f,0.0f,0.0f};
	State_GPU->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);

	InitializeVectorNeuronState_GPU2();

	//Initialize the array that stores the number of input current synapses for each neuron in the model
	InitializeCurrentSynapsis(N_neurons);
}



__global__ void LIFTimeDrivenModel_IS_GPU_C_INTERFACE_InitializeClassGPU2(LIFTimeDrivenModel_IS_GPU2 ** NeuronModel_GPU2,
		float e_exc, float e_inh, float e_leak, float v_thr, float c_m, float tau_exc, float tau_inh, float tau_ref, float g_leak,
		float tau_nmda, char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new LIFTimeDrivenModel_IS_GPU2(e_exc, e_inh, e_leak, v_thr, c_m,
		tau_exc, tau_inh, tau_ref, g_leak, tau_nmda, integrationName, N_neurons, Buffer_GPU);
	}
}


void LIFTimeDrivenModel_IS_GPU_C_INTERFACE::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(LIFTimeDrivenModel_IS_GPU2 **));

	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
//REVISAR
	HANDLE_ERROR(hipMemcpy(integrationNameGPU, &integration_method_GPU->name[0], 32 * 4, hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integration_method_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	LIFTimeDrivenModel_IS_GPU_C_INTERFACE_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2,e_exc, e_inh, e_leak, v_thr,
		c_m, tau_exc, tau_inh, tau_ref, g_leak, tau_nmda, integrationNameGPU, N_neurons, integration_method_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}


__global__ void initializeVectorNeuronState_GPU2(LIFTimeDrivenModel_IS_GPU2 ** NeuronModel_GPU2, int NumberOfVariables, float * InitialStateGPU, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(NumberOfVariables, InitialStateGPU, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}


void LIFTimeDrivenModel_IS_GPU_C_INTERFACE::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU_C_INTERFACE *state = (VectorNeuronState_GPU_C_INTERFACE *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->NumberOfVariables, state->InitialStateGPU, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(LIFTimeDrivenModel_IS_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2);
	}
}


void LIFTimeDrivenModel_IS_GPU_C_INTERFACE::DeleteClassGPU2(){
	if (NeuronModel_GPU2 != 0){
		DeleteClass_GPU2 << <1, 1 >> >(NeuronModel_GPU2);
		hipFree(NeuronModel_GPU2);
	}
}


__global__ void SetEnabledSynapsis_GPU2(LIFTimeDrivenModel_IS_GPU2 ** NeuronModel_GPU2, bool new_EXC, bool new_INH, bool new_NMDA, bool new_EXT_I){
	if (blockIdx.x == 0 && threadIdx.x == 0){
		(*NeuronModel_GPU2)->SetEnabledSynapsis(new_EXC, new_INH, new_NMDA, new_EXT_I);
	}
}


bool LIFTimeDrivenModel_IS_GPU_C_INTERFACE::CheckSynapseType(Interconnection * connection){
	int Type = connection->GetType();
	if (Type<N_TimeDependentNeuronState && Type >= 0){
		//activaty synapse type
		if (Type == 0 && EXC == false){
			EXC = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 1 && INH == false){
			INH = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 2 && NMDA == false){
			NMDA = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}
		if (Type == 3 && EXT_I == false){
			EXT_I = true;
			HANDLE_ERROR(hipSetDevice(GPUsIndex[GPU_index]));
			SetEnabledSynapsis_GPU2 << <1, 1 >> >(NeuronModel_GPU2, EXC, INH, NMDA, EXT_I);
		}

		NeuronModel * model = connection->GetSource()->GetNeuronModel();
		//Synapse types that process input spikes
		if (Type < N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_SPIKE){
				return true;
			}
			else{
			cout << "Synapses type " << Type << " of neuron model " << LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetName() << " must receive spikes. The source model generates currents." << endl;
				return false;
			}
		}
		//Synapse types that process input current
		if (Type == N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_CURRENT){
				connection->SetSubindexType(this->CurrentSynapsis->GetNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState()));
				this->CurrentSynapsis->IncrementNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState());
				return true;
			}
			else{
				cout << "Synapses type " << Type << " of neuron model " << LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetName() << " must receive current. The source model generates spikes." << endl;
				return false;
			}
		}
	}
	else{
		cout << "Neuron model " << LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetName() << " does not support input synapses of type " << Type << ". Just defined " << N_TimeDependentNeuronState << " synapses types." << endl;
		return false;
	}
}

std::map<std::string, boost::any> LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetParameters() const {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetParameters();
	newMap["e_exc"] = boost::any(this->e_exc); // Excitatory reversal potential (V)
	newMap["e_inh"] = boost::any(this->e_inh); // Inhibitory reversal potential (V)
	newMap["e_leak"] = boost::any(this->e_leak); // Effective leak potential (V)
	newMap["v_thr"] = boost::any(this->v_thr); // Effective threshold potential (V)
	newMap["c_m"] = boost::any(float(this->c_m)); // Membrane capacitance (F)
	newMap["tau_exc"] = boost::any(this->tau_exc); // AMPA (excitatory) receptor time constant (s)
	newMap["tau_inh"] = boost::any(this->tau_inh); // GABA (inhibitory) receptor time constant (s)
	newMap["tau_ref"] = boost::any(this->tau_ref); // Refractory period (s)
	newMap["g_leak"] = boost::any(float(this->g_leak)); // Leak conductance (S)
	newMap["tau_nmda"] = boost::any(this->tau_nmda); // NMDA (excitatory) receptor time constant (s)
	return newMap;
}

std::map<std::string, boost::any> LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetSpecificNeuronParameters(int index) const throw (EDLUTException){
	return GetParameters();
}

void LIFTimeDrivenModel_IS_GPU_C_INTERFACE::SetParameters(std::map<std::string, boost::any> param_map) throw (EDLUTException){

	// Search for the parameters in the dictionary
	std::map<std::string,boost::any>::iterator it=param_map.find("e_exc");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_exc = new_param;
		param_map.erase(it);
	}

	it=param_map.find("e_inh");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_inh = new_param;
		param_map.erase(it);
	}

	it=param_map.find("e_leak");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->e_leak = new_param;
		param_map.erase(it);
	}

	it = param_map.find("v_thr");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->v_thr = new_param;
		param_map.erase(it);
	}

	it=param_map.find("c_m");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->c_m = new_param;
		param_map.erase(it);
	}

	it=param_map.find("tau_exc");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_exc = new_param;
		param_map.erase(it);
	}

	it=param_map.find("tau_inh");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_inh = new_param;
		param_map.erase(it);
	}

	it=param_map.find("tau_ref");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_ref = new_param;
		param_map.erase(it);
	}

	it = param_map.find("g_leak");
	if (it != param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->g_leak = new_param;
		param_map.erase(it);
	}

	it=param_map.find("tau_nmda");
	if (it!=param_map.end()){
		float new_param = boost::any_cast<float>(it->second);
		this->tau_nmda = new_param;
		param_map.erase(it);
	}

	// Search for the parameters in the dictionary
	TimeDrivenNeuronModel_GPU_C_INTERFACE::SetParameters(param_map);
	return;
}


IntegrationMethod_GPU_C_INTERFACE * LIFTimeDrivenModel_IS_GPU_C_INTERFACE::CreateIntegrationMethod(ModelDescription imethodDescription) throw (EDLUTException){
	return IntegrationMethodFactory_GPU_C_INTERFACE<LIFTimeDrivenModel_IS_GPU_C_INTERFACE>::CreateIntegrationMethod_GPU(imethodDescription, (LIFTimeDrivenModel_IS_GPU_C_INTERFACE*) this);
}


std::map<std::string, boost::any> LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetDefaultParameters() {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetDefaultParameters<LIFTimeDrivenModel_IS_GPU_C_INTERFACE>();
	newMap["e_exc"] = boost::any(0.0f); // Excitatory reversal potential (V)
	newMap["e_inh"] = boost::any(-80.0e-3f); // Inhibitory reversal potential (V)
	newMap["e_leak"] = boost::any(-65.0e-3f); // Effective leak potential (V)
	newMap["v_thr"] = boost::any(-50.0e-3f); // Effective threshold potential (V)
	newMap["c_m"] = boost::any(2.0e-12f); // Membrane capacitance (F)
	newMap["tau_exc"] = boost::any(5.0e-3f); // AMPA (excitatory) receptor time constant (s)
	newMap["tau_inh"] = boost::any(10.0e-3f); // GABA (inhibitory) receptor time constant (s)
	newMap["tau_ref"] = boost::any(1.0e-3f); // Refractory period (s)
	newMap["g_leak"] = boost::any(0.2e-9f); // Leak conductance (S)
	newMap["tau_nmda"] = boost::any(20.0e-3f); // NMDA (excitatory) receptor time constant (s)
	return newMap;
}

NeuronModel* LIFTimeDrivenModel_IS_GPU_C_INTERFACE::CreateNeuronModel(ModelDescription nmDescription){
	LIFTimeDrivenModel_IS_GPU_C_INTERFACE * nmodel = new LIFTimeDrivenModel_IS_GPU_C_INTERFACE();
	nmodel->SetParameters(nmDescription.param_map);
	return nmodel;
}

ModelDescription LIFTimeDrivenModel_IS_GPU_C_INTERFACE::ParseNeuronModel(std::string FileName) throw (EDLUTFileException){
	FILE *fh;
	ModelDescription nmodel;
	nmodel.model_name = LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetName();
	long Currentline = 0L;
	fh = fopen(FileName.c_str(), "rt");
	if (!fh) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_NEURON_MODEL_OPEN, REPAIR_NEURON_MODEL_NAME, Currentline, FileName.c_str());
	}

	Currentline = 1L;
	skip_comments(fh, Currentline);

	float param;
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_E_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_E_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_E_LEAK, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_leak"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_V_THR, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["v_thr"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_C_M, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["c_m"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_TAU_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_TAU_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_TAU_REF, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_ref"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_G_LEAK, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["g_leak"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_IS_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_IS_TAU_NMDA, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_nmda"] = boost::any(param);

	skip_comments(fh, Currentline);
	try {
		ModelDescription intMethodDescription = TimeDrivenNeuronModel_GPU_C_INTERFACE::ParseIntegrationMethod<LIFTimeDrivenModel_IS_GPU_C_INTERFACE>(fh, Currentline);
		nmodel.param_map["int_meth"] = boost::any(intMethodDescription);
	}
	catch (EDLUTException exc) {
		throw EDLUTFileException(exc, Currentline, FileName.c_str());
	}

	nmodel.param_map["name"] = boost::any(LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetName());

	fclose(fh);

	return nmodel;
}

std::string LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetName(){
	return "LIFTimeDrivenModel_IS_GPU";
}

std::map<std::string, std::string> LIFTimeDrivenModel_IS_GPU_C_INTERFACE::GetNeuronModelInfo() {
	// Return a dictionary with the parameters
	std::map<std::string, std::string> newMap;
	newMap["info"] = std::string("GPU Time-driven Leaky Integrate and Fire (LIF) neuron model with one differential equations(membrane potential (v)) and four types of input synapses: AMPA (excitatory), GABA (inhibitory), NMDA (excitatory) and external input current (set on pA). THIS MODEL USES THE INTERNATIONAL SYSTEM UNITS IN ITS PARAMETERS.");
	newMap["e_exc"] = std::string("Excitatory reversal potential (V)");
	newMap["e_inh"] = std::string("Inhibitory reversal potential (V)");
	newMap["e_leak"] = std::string("Effective leak potential (V)");
	newMap["v_thr"] = std::string("Effective threshold potential (V)");
	newMap["c_m"] = std::string("Membrane capacitance (F)");
	newMap["tau_exc"] = std::string("AMPA (excitatory) receptor time constant (s)");
	newMap["tau_inh"] = std::string("GABA (inhibitory) receptor time constant (s)");
	newMap["tau_ref"] = std::string("Refractory period (s)");
	newMap["g_leak"] = std::string("Leak conductance (S)");
	newMap["tau_nmda"] = std::string("NMDA (excitatory) receptor time constant (s)");
	newMap["int_meth"] = std::string("Integraton method dictionary (from the list of available integration methods in GPU)");

	return newMap;
}