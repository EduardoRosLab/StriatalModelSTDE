#include "hip/hip_runtime.h"
/***************************************************************************
 *                           LIFTimeDrivenModel_1_3_GPU_C_INTERFACE.cu     *
 *                           -------------------                           *
 * copyright            : (C) 2012 by Francisco Naveros                    *
 * email                : fnaveros@ugr.es                                  *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/LIFTimeDrivenModel_1_3_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/LIFTimeDrivenModel_1_3_GPU2.cuh"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU_C_INTERFACE.cuh"
#include "../../include/neuron_model/CurrentSynapseModel.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include "hip/hip_runtime.h"
#include ""

#include "integration_method/IntegrationMethodFactory_GPU_C_INTERFACE.cuh"


LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::LIFTimeDrivenModel_1_3_GPU_C_INTERFACE() : TimeDrivenNeuronModel_GPU_C_INTERFACE(SecondScale), e_exc(0), e_inh(0), e_leak(0), g_leak(0), c_m(0), v_thr(0), tau_exc(0), tau_inh(0),
tau_ref(0),  NeuronModel_GPU2(0){
	std::map<std::string, boost::any> param_map = LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetDefaultParameters();
	param_map["name"] = LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetName();
	this->SetParameters(param_map);

	this->State = (VectorNeuronState_GPU_C_INTERFACE *) new VectorNeuronState_GPU_C_INTERFACE(N_NeuronStateVariables);
}

LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::~LIFTimeDrivenModel_1_3_GPU_C_INTERFACE(void){
	DeleteClassGPU2();
}

VectorNeuronState * LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::ProcessInputSpike(Interconnection * inter, double time){
	this->State_GPU->AuxStateCPU[inter->GetType()*State_GPU->GetSizeState() + inter->GetTargetNeuronModelIndex()] += inter->GetWeight();

	return 0;
}


__global__ void LIFTimeDrivenModel_1_3_GPU_C_INTERFACE_UpdateState(LIFTimeDrivenModel_1_3_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}

		
bool LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::UpdateState(int index, double CurrentTime){
	//update input current values from electrical coupling synapses.
	for (int i = 0; i < State_GPU->GetSizeState(); i++){
		this->State_GPU->AuxStateCPU[(N_TimeDependentNeuronState - 1)*State_GPU->GetSizeState() + i] = this->CurrentSynapses->GetTotalCurrent(i);
	}

	if(prop.canMapHostMemory){
		LIFTimeDrivenModel_1_3_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(State_GPU->AuxStateGPU,State_GPU->AuxStateCPU,this->N_TimeDependentNeuronState*State_GPU->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		LIFTimeDrivenModel_1_3_GPU_C_INTERFACE_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(State_GPU->InternalSpikeCPU,State_GPU->InternalSpikeGPU,State_GPU->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}
	
	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(State_GPU->VectorNeuronStates,State_GPU->VectorNeuronStates_GPU,State_GPU->GetNumberOfVariables()*State_GPU->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastUpdate,State_GPU->LastUpdateGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(State_GPU->LastSpikeTime,State_GPU->LastSpikeTimeGPU,State_GPU->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	} 

	HANDLE_ERROR(hipEventRecord(stop, 0)); 
	HANDLE_ERROR(hipEventSynchronize(stop));

	memset(State_GPU->AuxStateCPU,0,N_TimeDependentNeuronState*State_GPU->SizeStates*sizeof(float));

	return false;
}


enum NeuronModelOutputActivityType LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetModelOutputActivityType(){
	return OUTPUT_SPIKE;
}

enum NeuronModelInputActivityType LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetModelInputActivityType(){
	return INPUT_SPIKE_AND_CURRENT;
}


ostream & LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::PrintInfo(ostream & out){
	out << "- Leaky Time-Driven Model GPU: " << LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetName() << endl;
	out << "\tExcitatory reversal potential (e_exc): " << this->e_exc << "V" << endl;
	out << "\tInhibitory reversal potential (e_inh): " << this->e_inh << "V" << endl;
	out << "\tEffective leak potential (e_leak): " << this->e_leak << "V" << endl;
	out << "\tMembrane capacitance (c_m): " << this->c_m << "F" << endl;
	out << "\tEffective threshold potential (v_thr): " << this->v_thr << "V" << endl;
	out << "\tAMPA (excitatory) receptor time constant (tau_exc): " << this->tau_exc << "s" << endl;
	out << "\tGABA (inhibitory) receptor time constant (tau_inh): " << this->tau_inh << "s" << endl;
	out << "\tRefractory period (tau_ref): " << this->tau_ref << "s" << endl;
	out << "\tLeak conductance (g_leak): " << this->g_leak << "S" << endl;
	return out;
}	




void LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::InitializeStates(int N_neurons, int OpenMPQueueIndex){
	//Select the correnpondent device. 
	HANDLE_ERROR(hipSetDevice(GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));  
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));

	this->State_GPU = (VectorNeuronState_GPU_C_INTERFACE *) this->State;
	
	//Initialize neural state variables.
	float initialization[] = {e_leak,0.0f,0.0f,0.0f};
	State_GPU->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);

	InitializeVectorNeuronState_GPU2();

	//Initialize the array that stores the number of input current synapses for each neuron in the model
	this->CurrentSynapses = new CurrentSynapseModel(N_neurons);
}




__global__ void LIFTimeDrivenModel_1_3_GPU_C_INTERFACE_InitializeClassGPU2(LIFTimeDrivenModel_1_3_GPU2 ** NeuronModel_GPU2, 
		float e_exc, float e_inh, float e_leak, float v_thr, float c_m, float tau_exc, float tau_inh, float tau_ref, float g_leak, 
		char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new LIFTimeDrivenModel_1_3_GPU2(e_exc, e_inh, e_leak, 
			v_thr, c_m, tau_exc, tau_inh, tau_ref, g_leak, integrationName, N_neurons, Buffer_GPU);
	}
}


void LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(LIFTimeDrivenModel_1_3_GPU2 **));
	
	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
//REVISAR
	HANDLE_ERROR(hipMemcpy(integrationNameGPU, &integration_method_GPU->name[0], 32 * 4, hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integration_method_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	LIFTimeDrivenModel_1_3_GPU_C_INTERFACE_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2,e_exc, e_inh, e_leak, v_thr, 
		c_m, tau_exc, tau_inh, tau_ref, g_leak, integrationNameGPU, N_neurons, integration_method_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(LIFTimeDrivenModel_1_3_GPU2 ** NeuronModel_GPU2, int NumberOfVariables, float * InitialStateGPU, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(NumberOfVariables, InitialStateGPU, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}

void LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU_C_INTERFACE *state = (VectorNeuronState_GPU_C_INTERFACE *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->NumberOfVariables, state->InitialStateGPU, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(LIFTimeDrivenModel_1_3_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2); 
	}
}


void LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::DeleteClassGPU2(){
	if (NeuronModel_GPU2 != 0){
		DeleteClass_GPU2 << <1, 1 >> >(NeuronModel_GPU2);
		hipFree(NeuronModel_GPU2);
	}
}


bool LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::CheckSynapseType(Interconnection * connection){
	int Type = connection->GetType();

	if (Type < N_TimeDependentNeuronState && Type >= 0){
		NeuronModel * model = connection->GetSource()->GetNeuronModel();
		//Synapse types that process input spikes 
		if (Type < N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_SPIKE){
				return true;
			}
			else{
			cout << "Synapses type " << Type << " of neuron model " << LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetName() << " must receive spikes. The source model generates currents." << endl;
				return false;
			}
		}
		//Synapse types that process input current 
		if (Type == N_TimeDependentNeuronState - 1){
			if (model->GetModelOutputActivityType() == OUTPUT_CURRENT){
				connection->SetSubindexType(this->CurrentSynapses->GetNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState()));
				this->CurrentSynapses->IncrementNInputCurrentSynapsesPerNeuron(connection->GetTarget()->GetIndex_VectorNeuronState());
				return true;
			}
			else{
				cout << "Synapses type " << Type << " of neuron model " << LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetName() << " must receive current. The source model generates spikes." << endl;
				return false;
			}
		}
	}
	else{
		cout << "Neuron model " << LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetName() << " does not support input synapses of type " << Type << ". Just defined " << N_TimeDependentNeuronState << " synapses types." << endl;
		return false;
	}
}


std::map<std::string, boost::any> LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetParameters() const {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetParameters();
	newMap["e_exc"] = boost::any(this->e_exc);
	newMap["e_inh"] = boost::any(this->e_inh);
	newMap["e_leak"] = boost::any(this->e_leak);
	newMap["c_m"] = boost::any(float(this->c_m));
	newMap["v_thr"] = boost::any(this->v_thr);
	newMap["tau_exc"] = boost::any(this->tau_exc);
	newMap["tau_inh"] = boost::any(this->tau_inh);
	newMap["tau_ref"] = boost::any(this->tau_ref);
	newMap["g_leak"] = boost::any(float(this->g_leak));
	return newMap;
}

void LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::SetParameters(std::map<std::string, boost::any> param_map) throw (EDLUTException){

	// Search for the parameters in the dictionary
	std::map<std::string,boost::any>::iterator it=param_map.find("e_exc");
	if (it!=param_map.end()){
		float newe_exc = boost::any_cast<float>(it->second);
		this->e_exc = newe_exc;
		param_map.erase(it);
	}

	it=param_map.find("e_inh");
	if (it!=param_map.end()){
		float newe_inh = boost::any_cast<float>(it->second);
		this->e_inh = newe_inh;
		param_map.erase(it);
	}

	it=param_map.find("e_leak");
	if (it!=param_map.end()){
		float newe_leak = boost::any_cast<float>(it->second);
		this->e_leak = newe_leak;
		param_map.erase(it);
	}

	it=param_map.find("c_m");
	if (it!=param_map.end()){
		float newc_m = boost::any_cast<float>(it->second);
		this->c_m = newc_m;
		param_map.erase(it);
	}

	it=param_map.find("v_thr");
	if (it!=param_map.end()){
		float newv_thr = boost::any_cast<float>(it->second);
		this->v_thr = newv_thr;
		param_map.erase(it);
	}

	it=param_map.find("tau_exc");
	if (it!=param_map.end()){
		float newtau_exc = boost::any_cast<float>(it->second);
		this->tau_exc = newtau_exc;
		param_map.erase(it);
	}

	it=param_map.find("tau_inh");
	if (it!=param_map.end()){
		float newtau_inh = boost::any_cast<float>(it->second);
		this->tau_inh = newtau_inh;
		param_map.erase(it);
	}

	it=param_map.find("tau_ref");
	if (it!=param_map.end()){
		float newtau_ref = boost::any_cast<float>(it->second);
		this->tau_ref = newtau_ref;
		param_map.erase(it);
	}

	it = param_map.find("g_leak");
	if (it != param_map.end()){
		float newg_leak = boost::any_cast<float>(it->second);
		this->g_leak = newg_leak;
		param_map.erase(it);
	}

	// Search for the parameters in the dictionary
	TimeDrivenNeuronModel_GPU_C_INTERFACE::SetParameters(param_map);
	return;
}


IntegrationMethod_GPU_C_INTERFACE * LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::CreateIntegrationMethod(ModelDescription imethodDescription) throw (EDLUTException){
	return IntegrationMethodFactory_GPU_C_INTERFACE<LIFTimeDrivenModel_1_3_GPU_C_INTERFACE>::CreateIntegrationMethod_GPU(imethodDescription, (LIFTimeDrivenModel_1_3_GPU_C_INTERFACE*) this);
}


std::map<std::string, boost::any> LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetDefaultParameters() {
	// Return a dictionary with the parameters
	std::map<std::string, boost::any> newMap = TimeDrivenNeuronModel_GPU_C_INTERFACE::GetDefaultParameters<LIFTimeDrivenModel_1_3_GPU_C_INTERFACE>();
	newMap["e_exc"] = boost::any(0.0f);
	newMap["e_inh"] = boost::any(-80.0e-3f);
	newMap["e_leak"] = boost::any(-65.0f);
	newMap["c_m"] = boost::any(float(110.0e-9));
	newMap["v_thr"] = boost::any(-50.0e-3f);
	newMap["tau_exc"] = boost::any(5.0e-3f);
	newMap["tau_inh"] = boost::any(10.0e-3f);
	newMap["tau_ref"] = boost::any(1.0e-3f);
	newMap["g_leak"] = boost::any(float(10.0e-9));
	return newMap;
}

NeuronModel* LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::CreateNeuronModel(ModelDescription nmDescription){
	LIFTimeDrivenModel_1_3_GPU_C_INTERFACE * nmodel = new LIFTimeDrivenModel_1_3_GPU_C_INTERFACE();
	nmodel->SetParameters(nmDescription.param_map);
	return nmodel;
}

ModelDescription LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::ParseNeuronModel(std::string FileName) throw (EDLUTFileException){
	FILE *fh;
	ModelDescription nmodel;
	nmodel.model_name = LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetName();
	long Currentline = 0L;
	fh = fopen(FileName.c_str(), "rt");
	if (!fh) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_NEURON_MODEL_OPEN, REPAIR_NEURON_MODEL_NAME, Currentline, FileName.c_str());
	}

	Currentline = 1L;
	skip_comments(fh, Currentline);

	float param;
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_E_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_E_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_E_LEAK, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["e_leak"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_V_THR, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["v_thr"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1 || param <= 0.0f) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_C_M, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["c_m"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_TAU_EXC, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_exc"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_TAU_INH, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_inh"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_TAU_REF, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["tau_ref"] = boost::any(param);

	skip_comments(fh, Currentline);
	if (fscanf(fh, "%f", &param) != 1) {
		throw EDLUTFileException(TASK_LIF_TIME_DRIVEN_MODEL_1_3_GPU_LOAD, ERROR_LIF_TIME_DRIVEN_MODEL_1_3_G_LEAK, REPAIR_NEURON_MODEL_VALUES, Currentline, FileName.c_str());
	}
	nmodel.param_map["g_leak"] = boost::any(param);

	skip_comments(fh, Currentline);
	try {
		ModelDescription intMethodDescription = TimeDrivenNeuronModel_GPU_C_INTERFACE::ParseIntegrationMethod<LIFTimeDrivenModel_1_3_GPU_C_INTERFACE>(fh, Currentline);
		nmodel.param_map["integration_method"] = boost::any(intMethodDescription);
	}
	catch (EDLUTException exc) {
		throw EDLUTFileException(exc, Currentline, FileName.c_str());
	}

	nmodel.param_map["name"] = boost::any(LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetName());

	fclose(fh);

	return nmodel;
}

std::string LIFTimeDrivenModel_1_3_GPU_C_INTERFACE::GetName(){
	return "LIFTimeDrivenModel_1_3_GPU";
}
